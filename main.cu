#include "hip/hip_runtime.h"
#include <sdl.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include "creature.hh"
#include "gui.hh"
#include "header.hh"

using namespace std;

// Screen size
#define WIDTH 640
#define HEIGHT 480

// Blocks and threads
#define THREADS_PER_BLOCKS 2
#define BLOCKS 4

// List of creatures
vector<creature> creatures;

// Update all creatures in the simulation
void updateCreatures();

// Draw a circle on a bitmap based on this creature's position and radius
void drawCreature(bitmap* bmp, creature c);

int main(int argc, char** argv) {
  // Seed the random number generator
  srand(time(NULL));
  
  // Create a GUI window
  gui ui("Evolution Simulation", WIDTH, HEIGHT);
  
  // Start with the running flag set to true
  bool running = true;
  
  // Render everything using this bitmap
  bitmap bmp(WIDTH, HEIGHT);

  while(running) {
    // Update creature positions
    updateCreatures();

    // Darken the bitmap instead of clearing it to leave trails
    bmp.darken(0.92);

    // Draw creatures
    drawCreature(&bmp, creatures[i]);
    
    // Display the rendered frame
    ui.display(bmp);
  }
  
  return 0;
}

// Draw a circle at the given creature's position
// Uses method from http://groups.csail.mit.edu/graphics/classes/6.837/F98/Lecture6/circle.html
void drawCreature(bitmap* bmp, creature c) {
  // Index of the creature? Have a struct of creatures? how do we make this GPU compatible
  // int index = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

  double center_x = c.pos().x();
  double center_y = c.pos().y();
  double radius = c.radius();
  rgb32 border_color;

  // Checking creature's food source to determine border color
  if (c.food_source() == 1) {
    border_color = rgb32(255, 0, 0);
  }
  else {
    border_color = rgb32(0, 255, 0);
  }
  
  // Loop over points in the upper-right quad of the circle
  for(double x = 0; x <= radius*1.1; x++) {
    for(double y = 0; y <= radius*1.1; y++) {
      // Is this point within the circle's radius?
      double dist = sqrt(pow(x, 2) + pow(y, 2));
      if(dist < radius) {
        if (dist > radius - 5) {
          bmp->set(center_x + x + x_offset, center_y + y + y_offset, border_color);
          bmp->set(center_x + x + x_offset, center_y - y + y_offset, border_color);
          bmp->set(center_x - x + x_offset, center_y - y + y_offset, border_color);
          bmp->set(center_x - x + x_offset, center_y + y + y_offset, border_color);
        }
        else {
          // Set this point, along with the mirrored points in the other three quads
          bmp->set(center_x + x + x_offset, center_y + y + y_offset, c.color());
          bmp->set(center_x + x + x_offset, center_y - y + y_offset, c.color());
          bmp->set(center_x - x + x_offset, center_y - y + y_offset, c.color());
          bmp->set(center_x - x + x_offset, center_y + y + y_offset, c.color());
        }
      }
    }
  }
}
